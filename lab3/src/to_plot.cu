#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <float.h>
#include <string.h> // Added for memset

#define CSC(call)                                               \
    do {                                                        \
        hipError_t status = call;                              \
        if (status != hipSuccess) {                            \
            fprintf(stderr, "[ERROR CUDA] File: '%s'; Line: %i; Message: %s.\n", \
                    __FILE__, __LINE__, hipGetErrorString(status));   \
            exit(1);                                            \
        }                                                       \
    } while (0)

// Macro to measure kernel execution time
#define MEASURE_KERNEL_TIME(kernel_call, time_var)          \
    do {                                                    \
        hipEvent_t _start, _stop;                          \
        CSC(hipEventCreate(&_start));                      \
        CSC(hipEventCreate(&_stop));                       \
        CSC(hipEventRecord(_start));                       \
        kernel_call;                                        \
        CSC(hipEventRecord(_stop));                        \
        CSC(hipEventSynchronize(_stop));                   \
        float _elapsed_time;                                \
        CSC(hipEventElapsedTime(&_elapsed_time, _start, _stop)); \
        time_var += _elapsed_time;                          \
        CSC(hipEventDestroy(_start));                      \
        CSC(hipEventDestroy(_stop));                       \
    } while (0)

#define MAX_CLASSES 32

__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

// Constant memory for averages and inverse covariance matrices
__constant__ double const_avg_r[MAX_CLASSES];
__constant__ double const_avg_g[MAX_CLASSES];
__constant__ double const_avg_b[MAX_CLASSES];
__constant__ double const_inv_covariance_matrices[MAX_CLASSES][3][3];

// Device function to invert a 3x3 matrix (double precision)
__device__ void invert_3x3_matrix_double(const double *a, double *inv_a) {
    double det = a[0]*(a[4]*a[8] - a[5]*a[7]) - a[1]*(a[3]*a[8] - a[5]*a[6]) + a[2]*(a[3]*a[7] - a[4]*a[6]);

    if (fabs(det) < 1e-12) {
        // Matrix is singular, set inverse to identity matrix
        inv_a[0] = 1.0; inv_a[1] = 0.0; inv_a[2] = 0.0;
        inv_a[3] = 0.0; inv_a[4] = 1.0; inv_a[5] = 0.0;
        inv_a[6] = 0.0; inv_a[7] = 0.0; inv_a[8] = 1.0;
        return;
    }

    double inv_det = 1.0 / det;

    inv_a[0] = (a[4]*a[8] - a[5]*a[7]) * inv_det;
    inv_a[1] = (a[2]*a[7] - a[1]*a[8]) * inv_det;
    inv_a[2] = (a[1]*a[5] - a[2]*a[4]) * inv_det;

    inv_a[3] = (a[5]*a[6] - a[3]*a[8]) * inv_det;
    inv_a[4] = (a[0]*a[8] - a[2]*a[6]) * inv_det;
    inv_a[5] = (a[2]*a[3] - a[0]*a[5]) * inv_det;

    inv_a[6] = (a[3]*a[7] - a[4]*a[6]) * inv_det;
    inv_a[7] = (a[1]*a[6] - a[0]*a[7]) * inv_det;
    inv_a[8] = (a[0]*a[4] - a[1]*a[3]) * inv_det;
}

// Kernel to read sample pixel values
__global__ void read_sample_pixels(uchar4 *d_image, int w, int h, int total_npj, int *d_coordinates_flat, double3 *d_sample_pixels) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= total_npj)
        return;

    int x = d_coordinates_flat[tid * 2];
    int y = d_coordinates_flat[tid * 2 + 1];

    // Check if x and y are within image bounds
    if (x < 0 || x >= w || y < 0 || y >= h) {
        d_sample_pixels[tid] = make_double3(0.0, 0.0, 0.0);
    } else {
        uchar4 p = d_image[y * w + x];
        d_sample_pixels[tid] = make_double3((double)p.x, (double)p.y, (double)p.z);
    }
}

// Kernel to compute sums for means
__global__ void compute_sums(int total_npj, int *d_class_ids, double3 *d_sample_pixels, double *d_sums_r, double *d_sums_g, double *d_sums_b) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= total_npj)
        return;

    int class_id = d_class_ids[tid];

    double3 p = d_sample_pixels[tid];

    atomicAddDouble(&d_sums_r[class_id], p.x);
    atomicAddDouble(&d_sums_g[class_id], p.y);
    atomicAddDouble(&d_sums_b[class_id], p.z);
}

// Kernel to compute averages
__global__ void compute_averages(int nc, double *d_sums_r, double *d_sums_g, double *d_sums_b, int *d_npjs, double *d_avg_r, double *d_avg_g, double *d_avg_b) {
    int c = threadIdx.x + blockIdx.x * blockDim.x;

    if (c >= nc)
        return;

    int npj = d_npjs[c];

    if (npj > 0) {
        d_avg_r[c] = d_sums_r[c] / npj;
        d_avg_g[c] = d_sums_g[c] / npj;
        d_avg_b[c] = d_sums_b[c] / npj;
    } else {
        d_avg_r[c] = 0.0;
        d_avg_g[c] = 0.0;
        d_avg_b[c] = 0.0;
    }
}

// Kernel to compute covariance matrices
__global__ void compute_covariances(int total_npj, int *d_class_ids, double3 *d_sample_pixels, double *d_avg_r, double *d_avg_g, double *d_avg_b, double *d_covariance_matrices) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= total_npj)
        return;

    int class_id = d_class_ids[tid];

    double3 p = d_sample_pixels[tid];

    double3 avg;
    avg.x = d_avg_r[class_id];
    avg.y = d_avg_g[class_id];
    avg.z = d_avg_b[class_id];

    double3 diff;
    diff.x = p.x - avg.x;
    diff.y = p.y - avg.y;
    diff.z = p.z - avg.z;

    // Indices of covariance matrix elements:
    // [0 1 2]
    // [3 4 5]
    // [6 7 8]

    double *cov = &d_covariance_matrices[class_id * 9];

    atomicAddDouble(&cov[0], diff.x * diff.x); // Cxx
    atomicAddDouble(&cov[1], diff.x * diff.y); // Cxy
    atomicAddDouble(&cov[2], diff.x * diff.z); // Cxz
    atomicAddDouble(&cov[4], diff.y * diff.y); // Cyy
    atomicAddDouble(&cov[5], diff.y * diff.z); // Cyz
    atomicAddDouble(&cov[8], diff.z * diff.z); // Czz
}

// Kernel to finalize covariance matrices
__global__ void finalize_covariances(int nc, double *d_covariance_matrices, int *d_npjs) {
    int c = threadIdx.x + blockIdx.x * blockDim.x;

    if (c >= nc)
        return;

    int npj = d_npjs[c];

    if (npj > 1) {
        double inv_np1 = 1.0 / (npj - 1);

        double *cov = &d_covariance_matrices[c * 9];

        // Finalize covariance matrix
        cov[0] *= inv_np1; // Cxx
        cov[1] *= inv_np1; // Cxy
        cov[2] *= inv_np1; // Cxz
        cov[4] *= inv_np1; // Cyy
        cov[5] *= inv_np1; // Cyz
        cov[8] *= inv_np1; // Czz

        // Set symmetric elements
        cov[3] = cov[1]; // Cyx = Cxy
        cov[6] = cov[2]; // Czx = Cxz
        cov[7] = cov[5]; // Czy = Cyz
    } else {
        // If npj <= 1, set covariance matrix to identity
        double *cov = &d_covariance_matrices[c * 9];
        cov[0] = 1.0; cov[1] = 0.0; cov[2] = 0.0;
        cov[3] = 0.0; cov[4] = 1.0; cov[5] = 0.0;
        cov[6] = 0.0; cov[7] = 0.0; cov[8] = 1.0;
    }
}

// Kernel to invert covariance matrices
__global__ void invert_covariances(int nc, double *d_covariance_matrices, double *d_inverse_covariance_matrices) {
    int c = threadIdx.x + blockIdx.x * blockDim.x;

    if (c >= nc)
        return;

    double *cov = &d_covariance_matrices[c * 9];
    double *inv_cov = &d_inverse_covariance_matrices[c * 9];

    invert_3x3_matrix_double(cov, inv_cov);
}

// Main kernel to compute Mahalanobis distances and assign class labels
__global__ void classify_kernel(uchar4 *d_image, int w, int h, int nc) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int offsetx = blockDim.x * gridDim.x;
    int offsety = blockDim.y * gridDim.y;

    for (int y = idy; y < h; y += offsety) {
        for (int x = idx; x < w; x += offsetx) {
            uchar4 p_uchar = d_image[y * w + x];
            double3 p;
            p.x = (double)p_uchar.x;
            p.y = (double)p_uchar.y;
            p.z = (double)p_uchar.z;

            double min_m = DBL_MAX;
            int label_class_idx_int = -1;

            for (int c = 0; c < nc; c++) {
                double3 avg_j;
                avg_j.x = const_avg_r[c];
                avg_j.y = const_avg_g[c];
                avg_j.z = const_avg_b[c];

                double3 diff;
                diff.x = p.x - avg_j.x;
                diff.y = p.y - avg_j.y;
                diff.z = p.z - avg_j.z;

                const double (*inv_cov)[3] = const_inv_covariance_matrices[c];

                double3 temp;
                temp.x = inv_cov[0][0] * diff.x + inv_cov[0][1] * diff.y + inv_cov[0][2] * diff.z;
                temp.y = inv_cov[1][0] * diff.x + inv_cov[1][1] * diff.y + inv_cov[1][2] * diff.z;
                temp.z = inv_cov[2][0] * diff.x + inv_cov[2][1] * diff.y + inv_cov[2][2] * diff.z;

                double m = diff.x * temp.x + diff.y * temp.y + diff.z * temp.z;

                if (m < min_m) {
                    min_m = m;
                    label_class_idx_int = c;
                } else if (m == min_m && c < label_class_idx_int) {
                    label_class_idx_int = c;
                }
            }

            // Convert to unsigned char
            unsigned char label_class = (unsigned char)(label_class_idx_int); // Changed from static_cast to C-style cast

            // Set the output pixel alpha channel to the class label
            d_image[y * w + x].w = label_class;
        }
    }
}

int main() {
    int w, h, block_size_x, block_size_y, grid_size_x, grid_size_y;
    int nc; // Number of classes

    // set block_size_x
    scanf("%d", &block_size_x);
    // set block_size_y
    scanf("%d", &block_size_y);
    // set grid_size_x
    scanf("%d", &grid_size_x);
    // set grid_size_y
    scanf("%d", &grid_size_y);

    char inputFilepath[4095], outputFilepath[4095];

    // Reading input and output file paths with buffer size limits
    if (scanf("%4095s", inputFilepath) != 1) {
        fprintf(stderr, "Error reading input filepath.\n");
        return 1;
    }
    if (scanf("%4095s", outputFilepath) != 1) {
        fprintf(stderr, "Error reading output filepath.\n");
        return 1;
    }

    // Reading the number of classes
    if (scanf("%d", &nc) != 1) {
        fprintf(stderr, "Error reading number of classes.\n");
        return 1;
    }

    // Enforce maximum number of classes
    if (nc > MAX_CLASSES) {
        fprintf(stderr, "Number of classes (%d) exceeds MAX_CLASSES (%d).\n", nc, MAX_CLASSES);
        return 1;
    }
    if (nc <= 0) {
        fprintf(stderr, "Number of classes must be positive (%d).\n", nc);
        return 1;
    }

    // Memory allocation for the number of pixels in each class
    int *npjs = (int *)malloc(nc * sizeof(int));
    if (npjs == NULL) {
        fprintf(stderr, "Memory allocation error for npjs!\n");
        return 1;
    }

    // Arrays to store the coordinates of pixels (2D dynamic array)
    int **coordinates = (int **)malloc(nc * sizeof(int *));
    if (coordinates == NULL) {
        fprintf(stderr, "Memory allocation error for coordinates!\n");
        free(npjs);
        return 1;
    }

    // Initialize coordinates to NULL for safe freeing in case of errors
    for (int c = 0; c < nc; c++) {
        coordinates[c] = NULL;
    }

    // Reading data for each class
    for (int c = 0; c < nc; c++) {
        // Reading the number of pixels
        if (scanf("%d", &npjs[c]) != 1) {
            fprintf(stderr, "Error reading number of pixels for class %d.\n", c);
            // Free allocated memory before exiting
            for (int i = 0; i < nc; i++) {
                if (coordinates[i] != NULL)
                    free(coordinates[i]);
            }
            free(coordinates);
            free(npjs);
            return 1;
        }

        if (npjs[c] < 0) {
            fprintf(stderr, "Number of pixels for class %d cannot be negative.\n", c);
            // Free allocated memory before exiting
            for (int i = 0; i < nc; i++) {
                if (coordinates[i] != NULL)
                    free(coordinates[i]);
            }
            free(coordinates);
            free(npjs);
            return 1;
        }

        // Allocating memory to store coordinates (npjs[c] pairs of numbers)
        coordinates[c] = (int *)malloc(npjs[c] * 2 * sizeof(int));
        if (coordinates[c] == NULL) {
            fprintf(stderr, "Memory allocation error for class %d coordinates!\n", c);
            // Free previously allocated memory
            for (int i = 0; i < c; i++) {
                free(coordinates[i]);
            }
            free(coordinates);
            free(npjs);
            return 1;
        }

        // Reading the coordinates of pixels
        for (int p = 0; p < npjs[c]; p++) {
            if (scanf("%d %d", &coordinates[c][p * 2], &coordinates[c][p * 2 + 1]) != 2) {
                fprintf(stderr, "Error reading coordinates for class %d, pixel %d.\n", c, p);
                // Free allocated memory before exiting
                for (int i = 0; i <= c; i++) {
                    free(coordinates[i]);
                }
                free(coordinates);
                free(npjs);
                return 1;
            }
        }
    }

    // Open input file
    FILE *fp = fopen(inputFilepath, "rb");
    if (fp == NULL) {
        fprintf(stderr, "Error opening input file: %s\n", inputFilepath);
        // Free allocated memory before exiting
        for (int c = 0; c < nc; c++) {
            free(coordinates[c]);
        }
        free(coordinates);
        free(npjs);
        return 1;
    }

    // Read image dimensions
    if (fread(&w, sizeof(int), 1, fp) != 1) {
        fprintf(stderr, "Error reading image width from input file.\n");
        fclose(fp);
        // Free allocated memory before exiting
        for (int c = 0; c < nc; c++) {
            free(coordinates[c]);
        }
        free(coordinates);
        free(npjs);
        return 1;
    }
    if (fread(&h, sizeof(int), 1, fp) != 1) {
        fprintf(stderr, "Error reading image height from input file.\n");
        fclose(fp);
        // Free allocated memory before exiting
        for (int c = 0; c < nc; c++) {
            free(coordinates[c]);
        }
        free(coordinates);
        free(npjs);
        return 1;
    }

    // Allocate host memory for image data
    uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
    if (data == NULL) {
        fprintf(stderr, "Memory allocation error for image data!\n");
        fclose(fp);
        // Free allocated memory before exiting
        for (int c = 0; c < nc; c++) {
            free(coordinates[c]);
        }
        free(coordinates);
        free(npjs);
        return 1;
    }

    // Read image data
    size_t items_read = fread(data, sizeof(uchar4), w * h, fp);
    if (items_read != (size_t)(w * h)) {
        fprintf(stderr, "Error reading image data from input file.\n");
        fclose(fp);
        free(data);
        // Free allocated memory before exiting
        for (int c = 0; c < nc; c++) {
            free(coordinates[c]);
        }
        free(coordinates);
        free(npjs);
        return 1;
    }
    fclose(fp);

    // Allocate device memory for image data
    uchar4 *d_image;
    CSC(hipMalloc(&d_image, w * h * sizeof(uchar4)));
    CSC(hipMemcpy(d_image, data, w * h * sizeof(uchar4), hipMemcpyHostToDevice));

    // Prepare data for processing
    int total_npj = 0;
    int *offsets = (int *)malloc((nc + 1) * sizeof(int)); // offsets[0..nc]
    if (offsets == NULL) {
        fprintf(stderr, "Memory allocation error for offsets!\n");
        // Free allocated memory before exiting
        free(data);
        for (int c = 0; c < nc; c++) {
            free(coordinates[c]);
        }
        free(coordinates);
        free(npjs);
        hipFree(d_image);
        return 1;
    }
    offsets[0] = 0;
    for (int c = 0; c < nc; c++) {
        // Validate coordinates are within image bounds
        for (int p = 0; p < npjs[c]; p++) {
            int x = coordinates[c][p * 2];
            int y = coordinates[c][p * 2 + 1];
            if (x < 0 || x >= w || y < 0 || y >= h) {
                fprintf(stderr, "Invalid coordinates (%d, %d) for class %d, pixel %d.\n", x, y, c, p);
                // Free allocated memory before exiting
                free(offsets);
                free(data);
                for (int i = 0; i < nc; i++) {
                    free(coordinates[i]);
                }
                free(coordinates);
                free(npjs);
                hipFree(d_image);
                return 1;
            }
        }
        offsets[c + 1] = offsets[c] + npjs[c];
    }
    total_npj = offsets[nc]; // total number of sample pixels

    // Flatten coordinates
    int *coordinates_flat = (int *)malloc(total_npj * 2 * sizeof(int));
    if (coordinates_flat == NULL) {
        fprintf(stderr, "Memory allocation error for flattened coordinates!\n");
        // Free allocated memory before exiting
        free(offsets);
        free(data);
        for (int c = 0; c < nc; c++) {
            free(coordinates[c]);
        }
        free(coordinates);
        free(npjs);
        hipFree(d_image);
        return 1;
    }
    int idx = 0;
    for (int c = 0; c < nc; c++) {
        for (int p = 0; p < npjs[c]; p++) {
            coordinates_flat[idx * 2] = coordinates[c][p * 2];       // x
            coordinates_flat[idx * 2 + 1] = coordinates[c][p * 2 + 1]; // y
            idx++;
        }
        free(coordinates[c]);
    }
    free(coordinates);

    // Prepare class IDs
    int *class_ids = (int *)malloc(total_npj * sizeof(int));
    if (class_ids == NULL) {
        fprintf(stderr, "Memory allocation error for class_ids!\n");
        // Free allocated memory before exiting
        free(coordinates_flat);
        free(offsets);
        free(data);
        free(npjs);
        hipFree(d_image);
        return 1;
    }
    idx = 0;
    for (int c = 0; c < nc; c++) {
        for (int p = 0; p < npjs[c]; p++) {
            class_ids[idx] = c;
            idx++;
        }
    }

    // Allocate device memory and copy data
    int *d_npjs;
    CSC(hipMalloc(&d_npjs, nc * sizeof(int)));
    CSC(hipMemcpy(d_npjs, npjs, nc * sizeof(int), hipMemcpyHostToDevice));

    int *d_class_ids;
    CSC(hipMalloc(&d_class_ids, total_npj * sizeof(int)));
    CSC(hipMemcpy(d_class_ids, class_ids, total_npj * sizeof(int), hipMemcpyHostToDevice));

    int *d_coordinates_flat;
    CSC(hipMalloc(&d_coordinates_flat, total_npj * 2 * sizeof(int)));
    CSC(hipMemcpy(d_coordinates_flat, coordinates_flat, total_npj * 2 * sizeof(int), hipMemcpyHostToDevice));

    double3 *d_sample_pixels;
    CSC(hipMalloc(&d_sample_pixels, total_npj * sizeof(double3)));

    float total_kernel_time = 0.0f; // Variable to accumulate kernel execution times

    // Read sample pixels
    hipEventCreateTIME((read_sample_pixels<<<block_size_x, grid_size_x>>>(d_image, w, h, total_npj, d_coordinates_flat, d_sample_pixels)), total_kernel_time);

    // Compute sums
    double *d_sums_r, *d_sums_g, *d_sums_b;
    CSC(hipMalloc(&d_sums_r, nc * sizeof(double)));
    CSC(hipMalloc(&d_sums_g, nc * sizeof(double)));
    CSC(hipMalloc(&d_sums_b, nc * sizeof(double)));
    CSC(hipMemset(d_sums_r, 0, nc * sizeof(double))); // Correct initialization
    CSC(hipMemset(d_sums_g, 0, nc * sizeof(double)));
    CSC(hipMemset(d_sums_b, 0, nc * sizeof(double)));

    hipEventCreateTIME((compute_sums<<<block_size_x, grid_size_x>>>(total_npj, d_class_ids, d_sample_pixels, d_sums_r, d_sums_g, d_sums_b)), total_kernel_time);

    // Compute averages
    double *d_avg_r, *d_avg_g, *d_avg_b;
    CSC(hipMalloc(&d_avg_r, nc * sizeof(double)));
    CSC(hipMalloc(&d_avg_g, nc * sizeof(double)));
    CSC(hipMalloc(&d_avg_b, nc * sizeof(double)));

    hipEventCreateTIME((compute_averages<<<block_size_x, grid_size_x>>>(nc, d_sums_r, d_sums_g, d_sums_b, d_npjs, d_avg_r, d_avg_g, d_avg_b)), total_kernel_time);

    // Copy averages to host and then to constant memory
    double *h_avg_r = (double *)malloc(nc * sizeof(double));
    double *h_avg_g = (double *)malloc(nc * sizeof(double));
    double *h_avg_b = (double *)malloc(nc * sizeof(double));
    if (h_avg_r == NULL || h_avg_g == NULL || h_avg_b == NULL) {
        fprintf(stderr, "Memory allocation error for host averages!\n");
        // Free allocated memory before exiting
        free(class_ids);
        free(coordinates_flat);
        free(offsets);
        free(data);
        free(h_avg_r); free(h_avg_g); free(h_avg_b);
        hipFree(d_image); hipFree(d_npjs); hipFree(d_class_ids);
        hipFree(d_coordinates_flat); hipFree(d_sample_pixels);
        hipFree(d_sums_r); hipFree(d_sums_g); hipFree(d_sums_b);
        hipFree(d_avg_r); hipFree(d_avg_g); hipFree(d_avg_b);
        return 1;
    }
    CSC(hipMemcpy(h_avg_r, d_avg_r, nc * sizeof(double), hipMemcpyDeviceToHost));
    CSC(hipMemcpy(h_avg_g, d_avg_g, nc * sizeof(double), hipMemcpyDeviceToHost));
    CSC(hipMemcpy(h_avg_b, d_avg_b, nc * sizeof(double), hipMemcpyDeviceToHost));

    CSC(hipMemcpyToSymbol(HIP_SYMBOL(const_avg_r), h_avg_r, nc * sizeof(double)));
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(const_avg_g), h_avg_g, nc * sizeof(double)));
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(const_avg_b), h_avg_b, nc * sizeof(double)));

    free(h_avg_r);
    free(h_avg_g);
    free(h_avg_b);

    // Compute covariance matrices
    double *d_covariance_matrices;
    CSC(hipMalloc(&d_covariance_matrices, nc * 9 * sizeof(double)));
    CSC(hipMemset(d_covariance_matrices, 0, nc * 9 * sizeof(double))); // Correct initialization

    hipEventCreateTIME((compute_covariances<<<block_size_x, grid_size_x>>>(total_npj, d_class_ids, d_sample_pixels, d_avg_r, d_avg_g, d_avg_b, d_covariance_matrices)), total_kernel_time);

    // Finalize covariance matrices
    hipEventCreateTIME((finalize_covariances<<<block_size_x, grid_size_x>>>(nc, d_covariance_matrices, d_npjs)), total_kernel_time);

    // Invert covariance matrices
    double *d_inverse_covariance_matrices;
    CSC(hipMalloc(&d_inverse_covariance_matrices, nc * 9 * sizeof(double)));

    hipEventCreateTIME((invert_covariances<<<block_size_x, grid_size_x>>>(nc, d_covariance_matrices, d_inverse_covariance_matrices)), total_kernel_time);

    // Copy inverse covariance matrices to host and then to constant memory
    double *h_inverse_covariance_matrices = (double *)malloc(nc * 9 * sizeof(double));
    if (h_inverse_covariance_matrices == NULL) {
        fprintf(stderr, "Memory allocation error for host inverse covariance matrices!\n");
        // Free allocated memory before exiting
        free(class_ids);
        free(coordinates_flat);
        free(offsets);
        free(data);
        hipFree(d_image); hipFree(d_npjs); hipFree(d_class_ids);
        hipFree(d_coordinates_flat); hipFree(d_sample_pixels);
        hipFree(d_sums_r); hipFree(d_sums_g); hipFree(d_sums_b);
        hipFree(d_avg_r); hipFree(d_avg_g); hipFree(d_avg_b);
        hipFree(d_covariance_matrices); hipFree(d_inverse_covariance_matrices);
        return 1;
    }
    CSC(hipMemcpy(h_inverse_covariance_matrices, d_inverse_covariance_matrices, nc * 9 * sizeof(double), hipMemcpyDeviceToHost));

    double h_inv_cov_matrices[MAX_CLASSES][3][3];
    memset(h_inv_cov_matrices, 0, sizeof(h_inv_cov_matrices)); // Initialize to zero
    for (int c = 0; c < nc; c++) {
        double *src = &h_inverse_covariance_matrices[c * 9];
        h_inv_cov_matrices[c][0][0] = src[0];
        h_inv_cov_matrices[c][0][1] = src[1];
        h_inv_cov_matrices[c][0][2] = src[2];
        h_inv_cov_matrices[c][1][0] = src[3];
        h_inv_cov_matrices[c][1][1] = src[4];
        h_inv_cov_matrices[c][1][2] = src[5];
        h_inv_cov_matrices[c][2][0] = src[6];
        h_inv_cov_matrices[c][2][1] = src[7];
        h_inv_cov_matrices[c][2][2] = src[8];
    }
    CSC(hipMemcpyToSymbol(HIP_SYMBOL(const_inv_covariance_matrices), h_inv_cov_matrices, sizeof(double) * nc * 3 * 3));

    free(h_inverse_covariance_matrices);

    // Run classification kernel
    hipEventCreateTIME((classify_kernel<<<dim3(grid_size_x, grid_size_y), dim3(block_size_x, block_size_y)>>>(d_image, w, h, nc)), total_kernel_time);

    // Copy result back to host
    CSC(hipMemcpy(data, d_image, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));

    CSC(hipFree(d_image));

    // Open output file
    fp = fopen(outputFilepath, "wb");
    if (fp == NULL) {
        fprintf(stderr, "Error opening output file: %s\n", outputFilepath);
        // Free allocated memory before exiting
        free(data);
        free(class_ids);
        free(coordinates_flat);
        free(offsets);
        free(npjs);
        hipFree(d_npjs); hipFree(d_class_ids);
        hipFree(d_coordinates_flat); hipFree(d_sample_pixels);
        hipFree(d_sums_r); hipFree(d_sums_g); hipFree(d_sums_b);
        hipFree(d_avg_r); hipFree(d_avg_g); hipFree(d_avg_b);
        hipFree(d_covariance_matrices); hipFree(d_inverse_covariance_matrices);
        return 1;
    }

    // Write image dimensions
    if (fwrite(&w, sizeof(int), 1, fp) != 1) {
        fprintf(stderr, "Error writing image width to output file.\n");
        fclose(fp);
        // Free allocated memory before exiting
        free(data);
        free(class_ids);
        free(coordinates_flat);
        free(offsets);
        free(npjs);
        hipFree(d_npjs); hipFree(d_class_ids);
        hipFree(d_coordinates_flat); hipFree(d_sample_pixels);
        hipFree(d_sums_r); hipFree(d_sums_g); hipFree(d_sums_b);
        hipFree(d_avg_r); hipFree(d_avg_g); hipFree(d_avg_b);
        hipFree(d_covariance_matrices); hipFree(d_inverse_covariance_matrices);
        return 1;
    }
    if (fwrite(&h, sizeof(int), 1, fp) != 1) {
        fprintf(stderr, "Error writing image height to output file.\n");
        fclose(fp);
        // Free allocated memory before exiting
        free(data);
        free(class_ids);
        free(coordinates_flat);
        free(offsets);
        free(npjs);
        hipFree(d_npjs); hipFree(d_class_ids);
        hipFree(d_coordinates_flat); hipFree(d_sample_pixels);
        hipFree(d_sums_r); hipFree(d_sums_g); hipFree(d_sums_b);
        hipFree(d_avg_r); hipFree(d_avg_g); hipFree(d_avg_b);
        hipFree(d_covariance_matrices); hipFree(d_inverse_covariance_matrices);
        return 1;
    }

    // Write image data
    size_t items_written = fwrite(data, sizeof(uchar4), w * h, fp);
    if (items_written != (size_t)(w * h)) {
        fprintf(stderr, "Error writing image data to output file.\n");
        fclose(fp);
        // Free allocated memory before exiting
        free(data);
        free(class_ids);
        free(coordinates_flat);
        free(offsets);
        free(npjs);
        hipFree(d_npjs); hipFree(d_class_ids);
        hipFree(d_coordinates_flat); hipFree(d_sample_pixels);
        hipFree(d_sums_r); hipFree(d_sums_g); hipFree(d_sums_b);
        hipFree(d_avg_r); hipFree(d_avg_g); hipFree(d_avg_b);
        hipFree(d_covariance_matrices); hipFree(d_inverse_covariance_matrices);
        return 1;
    }
    fclose(fp);
    free(data);

    printf("CUDA execution time: <%f ms>\n", total_kernel_time);

    // Free allocated memory
    free(npjs);
    free(offsets);
    free(coordinates_flat);
    free(class_ids);

    CSC(hipFree(d_npjs));
    CSC(hipFree(d_class_ids));
    CSC(hipFree(d_coordinates_flat));
    CSC(hipFree(d_sample_pixels));
    CSC(hipFree(d_sums_r));
    CSC(hipFree(d_sums_g));
    CSC(hipFree(d_sums_b));
    CSC(hipFree(d_avg_r));
    CSC(hipFree(d_avg_g));
    CSC(hipFree(d_avg_b));
    CSC(hipFree(d_covariance_matrices));
    CSC(hipFree(d_inverse_covariance_matrices));

    return 0;
}
