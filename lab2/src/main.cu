#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CSC(call)                                               \
    do {                                                        \
        hipError_t status = call;                              \
        if (status != hipSuccess) {                            \
            fprintf(stderr, "[ERROR CUDA] File: '%s'; Line: %i; Message: %s.\n", \
                    __FILE__, __LINE__, hipGetErrorString(status));   \
            exit(1);                                            \
        }                                                       \
    } while (0)

__global__ void kernel(hipTextureObject_t tex, uchar4 *out, int w, int h) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
   	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
    int x, y;
    uchar4 p;
    for(y = idy; y < h; y += offsety
		for(x = idx; x < w; x += offsetx) {
            p = tex2D<uchar4>(tex, x / w, y / h);
            // YOUR CODE. TASK: Выделение контуров. Метод Робертса.
            // out[y * w + x] = // YOUR CODE
            out[y * w + x] = make_uchar4(255 - p.x, 255 - p.y, 255 - p.z, p.w);
        }
}

int main() {
    int w, h;

    char inputFilepath[1024], outputFilepath[1024];
    scanf("%1024s", inputFilepath);
    scanf("%1024s", outputFilepath);

    FILE *fp = fopen(inputFilepath, "rb");
 	fread(&w, sizeof(int), 1, fp);
	fread(&h, sizeof(int), 1, fp);
 	uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
    fread(data, sizeof(uchar4), w * h, fp);
    fclose(fp);

    hipArray *arr;
    hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
    CSC(hipMallocArray(&arr, &ch, w, h));
    CSC(hipMemcpy2DToArray(arr, 0, 0, data, w * sizeof(uchar4), w * sizeof(uchar4), h, hipMemcpyHostToDevice));

    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = arr;

    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = true;

    hipTextureObject_t tex = 0;
    CSC(hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL));

    uchar4 *dev_out;
	CSC(hipMalloc(&dev_out, sizeof(uchar4) * w * h));

    hipEvent_t start, stop;
    CSC(hipEventCreate(&start));
    CSC(hipEventCreate(&stop));

    CSC(hipEventRecord(start));
    kernel<<<dim3(16, 16), dim3(32, 32)>>>(tex, dev_out, w, h);
    CSC(hipEventRecord(stop));
    CSC(hipEventSynchronize(stop));
    CSC(hipGetLastError());

    CSC(hipMemcpy(data, dev_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));

	CSC(hipDestroyTextureObject(tex));
	CSC(hipFreeArray(arr));
	CSC(hipFree(dev_out));

    float t;
    CSC(hipEventElapsedTime(&t, start, stop));
    CSC(hipEventDestroy(start));
    CSC(hipEventDestroy(stop));

    printf("CUDA execution time: %f ms\n", t);

    fp = fopen(outputFilepath, "wb");
	fwrite(&w, sizeof(int), 1, fp);
	fwrite(&h, sizeof(int), 1, fp);
	fwrite(data, sizeof(uchar4), w * h, fp);
	fclose(fp);
    free(data);
    return 0;
}
