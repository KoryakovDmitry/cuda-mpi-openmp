#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define CSC(call)                                               \
    do {                                                        \
        hipError_t status = call;                              \
        if (status != hipSuccess) {                            \
            fprintf(stderr, "[ERROR CUDA] File: '%s'; Line: %i; Message: %s.\n", \
                    __FILE__, __LINE__, hipGetErrorString(status));   \
            exit(1);                                            \
        }                                                       \
    } while (0)

#define ALLOCATE_VECTOR_CPU(host_arr, n)                        \
    double *host_arr = (double *)malloc(n * sizeof(double));    \
    if (host_arr == NULL) {                                     \
        fprintf(stderr, "[ERROR CPU] File: '%s'; Line: %i; Error in allocating mem for vector: `%s`\n", __FILE__, __LINE__, #host_arr); \
        return 1;                                               \
    }

__global__ void kernel(double *arr_1, double *arr_2, double *arr_3, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockDim.x * gridDim.x;
    while (idx < n) {
        arr_3[idx] = arr_1[idx] - arr_2[idx];
        idx += offset;
    }
}

int main() {

    int n, i;

    // set dim of vector
    scanf("%d", &n);

    // set mem for first vector on CPU
    ALLOCATE_VECTOR_CPU(host_arr_1, n);
    // set mem for second vector on CPU
    ALLOCATE_VECTOR_CPU(host_arr_2, n);
    // set mem for result vector on CPU
    ALLOCATE_VECTOR_CPU(host_arr_3, n);

    // fill first vector
    for (i = 0; i < n; i++) {
        scanf("%lf", &host_arr_1[i]);
    }
    // fill second vector
    for (i = 0; i < n; i++) {
        scanf("%lf", &host_arr_2[i]);
    }

    double *dev_arr_1, *dev_arr_2, *dev_arr_3;

    CSC(hipMalloc(&dev_arr_1, sizeof(double) * n));
    CSC(hipMalloc(&dev_arr_2, sizeof(double) * n));
    CSC(hipMalloc(&dev_arr_3, sizeof(double) * n));

    CSC(hipMemcpy(dev_arr_1, host_arr_1, sizeof(double) * n, hipMemcpyHostToDevice));
    CSC(hipMemcpy(dev_arr_2, host_arr_2, sizeof(double) * n, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    CSC(hipEventCreate(&start));
    CSC(hipEventCreate(&stop));

    CSC(hipEventRecord(start));
    kernel<<<512, 512>>>(dev_arr_1, dev_arr_2, dev_arr_3, n);
    CSC(hipEventRecord(stop));
    CSC(hipEventSynchronize(stop));
    CSC(hipGetLastError());

    float t;
    CSC(hipEventElapsedTime(&t, start, stop));
    CSC(hipEventDestroy(start));
    CSC(hipEventDestroy(stop));

    // printf("CUDA execution time: %f ms\n", t);

    CSC(hipMemcpy(host_arr_3, dev_arr_3, sizeof(double) * n, hipMemcpyDeviceToHost));

    for (i = 0; i < n; i++) {
        printf("%.10e ", host_arr_3[i]);
    }
    // printf("\n");

    CSC(hipFree(dev_arr_1));
    CSC(hipFree(dev_arr_2));
    CSC(hipFree(dev_arr_3));
    free(host_arr_1);
    free(host_arr_2);
    free(host_arr_3);

    return 0;
}
